#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <array>
#include <fstream>
#include <chrono>
#include <algorithm>

// parameters
const double G = 6.67e-11;
const int N_BODIES = 1000;
const int N_DIM = 3;
// e.g. if N_BODIES > 896 and N_DIM > 9, the gpu breaks becasue of register preassure, other way around is fine
const double DELTA_T = 1.0;
const int N_SIMULATIONS = 100;
 //  large numbers of bodies, dimensions and simulation steps introduce accumulated miscalculations of small numbers
const double LOWER_M = 1e-6;
const double HIGHER_M = 1e6;
const double LOWER_P = -1e-1;
const double HIGHER_P = 1e-1;
const double LOWER_V = -1e-4;
const double HIGHER_V = 1e-4;

const int MAX_BLOCK_SIZE = 1024; // limit for threads in CUDA

// structures
using Vector = std::array<double, N_DIM>;
using Positions = std::array<Vector, N_BODIES>;
using Velocities = std::array<Vector, N_BODIES>;
using Forces = std::array<Vector, N_BODIES>;
using Masses = std::array<double, N_BODIES>;
using Accelerations = std::array<Vector, N_BODIES>;

double generateRandom(double lower, double upper) {
    return lower + static_cast<double>(std::rand()) / RAND_MAX * (upper - lower);
}

double generateLogRandom(double lower, double upper) {
    return std::pow(10, std::log10(lower) + static_cast<double>(std::rand()) / RAND_MAX * (std::log10(upper) - std::log10(lower)));
}

void initializeMasses(Masses& masses, double LOWER_M, double HIGHER_M) {
    for (double& mass : masses) {
        mass = generateLogRandom(LOWER_M, HIGHER_M);
    }
}

void initializeVectors(Positions& vectors, double lower, double upper) {
    for (auto& vector : vectors) {
        for (double& component : vector) {
            component = generateRandom(lower, upper);
        }
    }
}

void computeForces(const Positions& positions, const Masses& masses, Forces& forces) {
    for (int i = 0; i < N_BODIES; ++i) {
        Vector sum = {};
        for (int j = 0; j < N_BODIES; ++j) {
            if (i == j) continue;

            double distance_squared = 0.0;
            Vector displacement = {};
            for (int k = 0; k < N_DIM; ++k) {
                displacement[k] = positions[j][k] - positions[i][k];
                distance_squared += displacement[k] * displacement[k];
            }

            double distance = std::sqrt(distance_squared);
            double factor = G * masses[i] * masses[j] / (distance_squared * distance);

            for (int k = 0; k < N_DIM; ++k) {
                sum[k] += factor * displacement[k];
            }
        }
        forces[i] = sum;
    }
}

__global__ void computeForcesGpu(double* positions, double* masses, double* forces) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N_BODIES)
        return;
    
    double sum[N_DIM] = {};
    for (int j = 0; j < N_BODIES; ++j) {
        if (idx == j) continue;

        double distance_squared = 0.0;
        double displacement[N_DIM] = {};
        for (int k = 0; k < N_DIM; ++k) {
            displacement[k] = positions[j * N_DIM + k] - positions[idx * N_DIM + k];
            distance_squared += displacement[k] * displacement[k];
        }

        double distance = std::sqrt(distance_squared);
        double factor = G * masses[idx] * masses[j] / (distance_squared * distance);

        for (int k = 0; k < N_DIM; ++k) {
            sum[k] += factor * displacement[k];
        }
    }
    for (int k = 0; k < N_DIM; ++k) {
        forces[idx * N_DIM + k] = sum[k];
    }
}

void updateAccelerations(const Forces& forces, const Masses& masses, Positions& accelerations) {
    for (int i = 0; i < N_BODIES; ++i) {
        for (int k = 0; k < N_DIM; ++k) {
            accelerations[i][k] = forces[i][k] / masses[i];
        }
    }
}

__global__ void updateAccelerationsGpu(double* forces, double* masses, double* accelerations) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N_BODIES)
        return;

    for (int k = 0; k < N_DIM; ++k) {
        accelerations[idx * N_DIM + k] = forces[idx * N_DIM + k] / masses[idx];
    }
}

void updateVelocities(Velocities& velocities, const Positions& accelerations, double DELTA_T) {
    for (int i = 0; i < N_BODIES; ++i) {
        for (int k = 0; k < N_DIM; ++k) {
            velocities[i][k] += accelerations[i][k] * DELTA_T;
        }
    }
}

__global__ void updateVelocitiesGpu(double* velocities, double* accelerations, double DELTA_T) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N_BODIES)
        return;
    
    for (int k = 0; k < N_DIM; ++k) {
        velocities[idx * N_DIM + k] += accelerations[idx * N_DIM + k] * DELTA_T;
    }
}

void updatePositions(Positions& positions, const Velocities& velocities, double DELTA_T) {
    for (int i = 0; i < N_BODIES; ++i) {
        for (int k = 0; k < N_DIM; ++k) {
            positions[i][k] += velocities[i][k] * DELTA_T;
        }
    }
}

__global__ void updatePositionsGpu(double* positions, double* velocities, double DELTA_T) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N_BODIES)
        return;
    
    for (int k = 0; k < N_DIM; ++k) {
        positions[idx * N_DIM + k] += velocities[idx * N_DIM + k] * DELTA_T;
    }
}

void printBodies(const Masses& masses, const Positions& positions, const Velocities& velocities) {
    for (int i = 0; i < N_BODIES; ++i) {
        std::cout << "Body " << i << ":\n";
        std::cout << "  Mass: " << masses[i] << "\n";
        std::cout << "  Position: [ ";
        for (const double& pos : positions[i]) {
            std::cout << pos << ' ';
        }
        std::cout << "]\n";
        std::cout << "  Velocity: [ ";
        for (const double& vel : velocities[i]) {
            std::cout << vel << ' ';
        }
        std::cout << "]\n";
    }
}

void savePositions(std::string& output_str, const Positions& positions, double time) {
    for (int i = 0; i < N_BODIES; ++i) {
        output_str += std::to_string(time) + " " + std::to_string(i) + " ";
        for (const double& pos : positions[i]) {
            output_str += std::to_string(pos) + " ";
        }
        output_str += "\n";
    }
}

void runSimulationCpu(Masses masses, Positions& positions, Velocities velocities) {
    Accelerations accelerations = {};
    Forces forces = {};

    std::ofstream positions_file("positions.txt");
    std::string output_str;

    double absolute_t = 0.0;
    savePositions(output_str, positions, absolute_t);
    //printBodies(masses, positions, velocities);
    
    for (int step = 0; step < N_SIMULATIONS; ++step) {
        absolute_t += DELTA_T;

        computeForces(positions, masses, forces);
        updateAccelerations(forces, masses, accelerations);
        updateVelocities(velocities, accelerations, DELTA_T);
        updatePositions(positions, velocities, DELTA_T);
        savePositions(output_str, positions, absolute_t);
    }
    
    positions_file << output_str;
    positions_file.close();
}

void runSimulationGpu(Masses masses, Positions& positions, Velocities velocities) {
    double* masses_d;
    double* positions_d;
    double* velocities_d;
    double* accelerations_d;
    double* forces_d;

    hipMalloc( (void**)&masses_d, N_BODIES * sizeof(double));
    hipMalloc( (void**)&positions_d, N_BODIES * N_DIM * sizeof(double));
    hipMalloc( (void**)&velocities_d, N_BODIES * N_DIM * sizeof(double));
    hipMalloc( (void**)&accelerations_d, N_BODIES * N_DIM * sizeof(double));
    hipMalloc( (void**)&forces_d, N_BODIES * N_DIM * sizeof(double));

    hipMemcpy( masses_d, masses.data(), N_BODIES * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy( positions_d, positions.data(), N_BODIES * N_DIM * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy( velocities_d, velocities.data(), N_BODIES * N_DIM * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = (N_BODIES <= MAX_BLOCK_SIZE) ? N_BODIES : MAX_BLOCK_SIZE;

    dim3 dimBlock(blockSize);
	dim3 dimGrid((N_BODIES + blockSize - 1) / blockSize);

    double absolute_t = 0.0;
    
    for (int step = 0; step < N_SIMULATIONS; ++step) {
        absolute_t += DELTA_T;

        computeForcesGpu<<<dimGrid, dimBlock>>>(positions_d, masses_d, forces_d);
        hipDeviceSynchronize();
        updateAccelerationsGpu<<<dimGrid, dimBlock>>>(forces_d, masses_d, accelerations_d);
        hipDeviceSynchronize();
        updateVelocitiesGpu<<<dimGrid, dimBlock>>>(velocities_d, accelerations_d, DELTA_T);
        hipDeviceSynchronize();
        updatePositionsGpu<<<dimGrid, dimBlock>>>(positions_d, velocities_d, DELTA_T);
        hipDeviceSynchronize();
    }
    
    hipMemcpy( positions.data(), positions_d, N_BODIES * N_DIM * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(masses_d);
    hipFree(positions_d);
    hipFree(velocities_d);
    hipFree(accelerations_d);
    hipFree(forces_d);
}

void checkEqual(const auto& first, const auto& second, const std::string& name) {
    bool allEqual = true;

    for (size_t i = 0; i < first.size(); ++i) {
        for (size_t j = 0; j < first[i].size(); ++j) {
            if (std::fabs(first[i][j] - second[i][j]) > 1e-3) {
                allEqual = false;
                std::cout << "Difference at index [" << i << "][" << j << "]: "
                          << "first = " << first[i][j]
                          << ", second = " << second[i][j]
                          << " , and the diff is: " << std::fabs(first[i][j] - second[i][j]) << std::endl;
                break;
            }
        }
    }
    if (allEqual) {
        std::cout << "\nThe " << name << " are the same.";
    } else {
        std::cout << "\n\n!!!!! The " << name << " are NOT the same !!!!!\n\n";
    }
}

int main() {
    std::srand(static_cast<unsigned>(std::time(0)));

    // structures
    Masses masses;
    Positions positions;
    Velocities velocities;

    // initialization
    initializeMasses(masses, LOWER_M, HIGHER_M);
    initializeVectors(positions, LOWER_P, HIGHER_P);
    initializeVectors(velocities, LOWER_V, HIGHER_V);

    // cpu simulation run

    Positions positions_cpu = positions;

    auto start_cpu = std::chrono::high_resolution_clock::now();

    runSimulationCpu(masses, positions_cpu, velocities);

    auto end_cpu = std::chrono::high_resolution_clock::now();
    auto duration_cpu = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu - start_cpu);

    // gpu simulation run

    Positions positions_gpu = positions;

    auto start_gpu = std::chrono::high_resolution_clock::now();

    runSimulationGpu(masses, positions_gpu, velocities);
    
    auto end_gpu = std::chrono::high_resolution_clock::now();
    auto duration_gpu = std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu);

    std::cout<<std::endl<<std::endl;

    checkEqual(positions_cpu, positions_gpu, "final positions");

    std::cout<<std::endl<<std::endl;

    std::cout << "CPU computation took " << duration_cpu.count() << " milliseconds." << std::endl;
    std::cout << "GPU computation took " << duration_gpu.count() << " milliseconds." << std::endl;

    std::cout<<std::endl<<std::endl;

    return 0;
}
